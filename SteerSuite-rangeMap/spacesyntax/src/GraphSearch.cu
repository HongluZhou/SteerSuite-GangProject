#include "hip/hip_runtime.h"
#define BLOCK_SIZE 256
#define MAX_DEPTH 5
#include <math.h>
#include <iostream>
#include "Structs.h"
#include "hip/hip_runtime.h"
#include <>
#include "GraphSearch.h"
#include <time.h>
#include <vector>

using namespace std;

bool selectBestGPU()
{
	int numDevices;
	hipError_t cudaResultCode = hipGetDeviceCount(&numDevices);
	if (cudaResultCode != hipSuccess){
		return false;
	}

	int maxMultiProc = 0;
	for (int i = 0; i < numDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		if (prop.multiProcessorCount > maxMultiProc) {
			maxMultiProc = prop.multiProcessorCount;
			hipSetDevice(i);
		}
	}

	return true;
}

__device__ unsigned return_thread_index()
{
	unsigned thread_num_in_block = threadIdx.x + (threadIdx.y * blockDim.x);
	unsigned block_num_in_grid = blockIdx.x + (blockIdx.y * gridDim.x);
	return (block_num_in_grid * blockDim.x * blockDim.y) + thread_num_in_block;
}

__device__ float solve_line_z(float xTarget, float x1, float z1, float tangent)
{
	return tangent*(xTarget - x1) + z1;
}

__device__ float solve_line_x(float zTarget, float x1, float z1, float tangent)
{
	return (zTarget - z1) / tangent + x1;
}


__device__ void rotate_axis(float& x1r, float& z1r, float tilt, float x1, float z1)
{
	float sinVal, cosVal;
	sinVal = sinf(tilt);
	cosVal = cosf(tilt);
	z1r = cosVal*z1 - sinVal*x1;
	x1r = sinVal*z1 + cosVal*x1;
}

__device__ int ray_cast(Point_Struct p1, Point_Struct p2, Obstacle_Struct* obstacles, int numObstacles)
{
	for (int i = 0; i < numObstacles; i++)
	{
		Obstacle_Struct obstacle = obstacles[i];
		float x1, z1, x2, z2, target;
		if (obstacle.type == SHADOW_OBSTACLE)
			continue;

		x1 = obstacle.pMin.x;
		z1 = obstacle.pMin.z;
		x2 = obstacle.pMax.x;
		z2 = obstacle.pMax.z;

		// Do axis rotation to get axis aligned obstacle
		float tilt, x1r, z1r, x2r, z2r, px1r, pz1r, px2r, pz2r, tangent;
		tilt = obstacle.tiltDegree;
		rotate_axis(x1r, z1r, tilt, x1, z1);
		rotate_axis(x2r, z2r, tilt, x2, z2);
		rotate_axis(px1r, pz1r, tilt, p1.x, p1.z);
		rotate_axis(px2r, pz2r, tilt, p2.x, p2.z);
		tangent = (pz2r - pz1r) / (px2r - px1r);

		float xpMin = (px1r < px2r) ? px1r : px2r;
		float xpMax = (px1r > px2r) ? px1r : px2r;
		float zpMin = (pz1r < pz2r) ? pz1r : pz2r;
		float zpMax = (pz1r > pz2r) ? pz1r : pz2r;

		if (x1r > x2r)
		{
			float tmp = x1r;
			x1r = x2r;
			x2r = tmp;
		}
		if (z1r > z2r)
		{
			float tmp = z1r;
			z1r = z2r;
			z2r = tmp;
		}

		// check if the obstacle is within points
		if (xpMin > x2r || xpMax < x1r || zpMin > z2r || zpMax < z1r)
			continue;

		// check if any of the points are inside the obstacle
		if (px1r >= x1r && px1r <= x2r && pz1r >= z1r && pz1r <= z2r)
			return obstacle.type;
		if (px2r >= x1r && px2r <= x2r && pz2r >= z1r && pz2r <= z2r)
			return obstacle.type;

		if (x1r >= xpMin && x1r <= xpMax)
		{
			target = solve_line_z(x1r, px1r, pz1r, tangent);
			if (target >= z1r && target <= z2r)
				if (target >= zpMin && target <= zpMax)
					return obstacle.type;
		}

		if (x2r >= xpMin && x2r <= xpMax)
		{
			target = solve_line_z(x2r, px1r, pz1r, tangent);
			if (target >= z1r && target <= z2r)
				if (target >= zpMin && target <= zpMax)
					return obstacle.type;
		}
		if (z1r >= zpMin && z1r <= zpMax)
		{
			target = solve_line_x(z1r, px1r, pz1r, tangent);
			if (target >= x1r && target <= x2r)
				if (target >= xpMin && target <= xpMax)
					return obstacle.type;
		}

	}
	return SHADOW_OBSTACLE;
}

__device__ bool hasLineOfSight(Point_Struct point1, Point_Struct point2, Obstacle_Struct* obstacles, int numObstacles)
{
	if (ray_cast(point1, point2, obstacles, numObstacles))
		return false;
	else
		return true;
}


__global__ void cuda_generate_graph(Point_Struct* grid, int* adjMatrix_ptr, int totalNodes, Obstacle_Struct* obstacles, int numObstacles)
{
	int threadIndex = return_thread_index();
	Point_Struct pt1 = grid[threadIndex];
	if (threadIndex < totalNodes)
	{
		for (int i = 0; i < totalNodes; i++)
		{
			Point_Struct pt2 = grid[i];
			if (hasLineOfSight(pt1, pt2, obstacles, numObstacles))
			{
				int index = totalNodes * threadIndex + i;
				adjMatrix_ptr[index] = 1;
			}
		}
	}
}

__global__ void cuda_generate_graph_mv1(Point_Struct* grid, char* adjMatrix_ptr, int totalNodes, Obstacle_Struct* obstacles, int numObstacles, char obsType)
{
	unsigned threadIndex = return_thread_index();
	unsigned threadIndexY = threadIndex / totalNodes;
	unsigned threadIndexX = threadIndex % totalNodes;
	Point_Struct pt1 = grid[threadIndexY];
	Point_Struct pt2 = grid[threadIndexX];
	if (threadIndexY <= threadIndexX)
	{
		unsigned index = totalNodes * threadIndexY + threadIndexX;
		unsigned index2 = totalNodes * threadIndexX + threadIndexY;
		if (adjMatrix_ptr[index] == STATIC_OBSTACLE) // static obstacle has blocked the path
			return;
		if (hasLineOfSight(pt1, pt2, obstacles, numObstacles))
		{
			// if there is no block, set matrix element to 0 (connected)
			adjMatrix_ptr[index] = 0; //use symmetry
			adjMatrix_ptr[index2] = 0;
		}
		else
		{
			// if there is an obstacle blocking the line of sight, set matrix element to obsType (1-2)
			adjMatrix_ptr[index] = obsType; //use symmetry
			adjMatrix_ptr[index2] = obsType;
		}
	}
}

__global__ void cuda_sequence_list(bool* frontMat, int* seqFront, int totalRef, int totalQuery)
{
	unsigned tid, count, ntree;
	count = 0;
	tid = return_thread_index();
	if (tid >= totalQuery)
		return;
	//ntree = tid % totalNodes;
	ntree = tid;
	for (unsigned s = 0; s < totalRef; s++)
	{
		if (frontMat[s + ntree * totalRef])
		{
			seqFront[count + ntree * totalRef] = s;
			count++;
		}
	}
}

__global__ void cuda_expand_forest(bool* parentMat, bool* childMat, int* seqFront, int* depthMat, Point_Struct* nodeList, char* adjMatrix, int depth, int totalNodes, int totalRef, int totalQuery)
{
	int tid, ntree, idX, idY;
	ntree = blockIdx.y;
	if (ntree >= totalQuery)
		return;
	//if (!nodeList[ntree].isQ)
	//	return;

	idX = threadIdx.x + (threadIdx.y * blockDim.x) + (blockIdx.x * BLOCK_SIZE);
	tid = threadIdx.x + (threadIdx.y * blockDim.x);
	if (idX >= totalRef)
		return;

	//extern __shared__ int seqf[];
	__shared__ int frontDepth;
	if (tid == 0)
	{
		if (depth == 0)
			frontDepth = 1;
		else
			frontDepth = depthMat[depth - 1 + ntree * MAX_DEPTH];
	}
	/*
	for (unsigned s = 0; s < totalNodes; s += BLOCK_SIZE)
	{
	if (tid + s < totalNodes)
	{
	seqf[tid + s] = seqFront[tid + s + ntree * totalNodes];
	}
	}*/
	__syncthreads();

	//if (!nodeList[idX].isRef)
	//	return;
	for (unsigned s = 0; s < frontDepth; s++)
	{
		if (parentMat[idX + ntree * totalRef])
			return;
		//idY = seqf[s];
		idY = seqFront[s + ntree * totalRef];
		//if (idY < 0)
		//	return;
		if (!adjMatrix[idX + idY * totalNodes]) // 0 is connected
		{
			childMat[idX + ntree * totalRef] = true;
			parentMat[idX + ntree * totalRef] = true;
		}
	}
}

__global__ void cuda_init_forest(bool* parentMat, bool* childMat, bool* frontMat, int* seqFront, int* depthMat, int totalRef, int totalQuery, int refOnly)
{
	int tid;
	tid = return_thread_index();
	if (tid >= totalQuery)
		return;
	seqFront[totalRef * tid] = tid + refOnly;
	/*
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalNodes;
	idY = tid / totalNodes;

	if (idY >= totalNodes)
		return;

	//childMat[idX + idY * totalNodes] = false;
	//seqFront[idX + idY * totalNodes] = -1;
	if (idX == idY)
	{
		frontMat[idX + idY * totalNodes] = true;
		//parentMat[idX + idY * totalNodes] = true;
	}
	else
	{
		frontMat[idX + idY * totalNodes] = false;
		//parentMat[idX + idY * totalNodes] = false;
	}

	//if (idX < MAX_DEPTH)
	//{
	//	depthMat[idX + idY * MAX_DEPTH] = 0;
	//}*/
}

__global__ void cuda_update_forest(bool* parentMat, bool* childMat, bool* frontMat, int* seqFront, int* depthMat, int depth, int totalRef, int totalQuery)
{
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalRef;
	idY = tid / totalRef;
	extern __shared__ int sumVal[];

	if (idY >= totalQuery)
		return;

	// Init shared memory (only where needed)
	if (sumVal[idY] != 0)
		sumVal[idY] = 0;
	__syncthreads();


	if (childMat[idX + idY * totalRef])
		atomicAdd(&sumVal[idY], 1);
	__syncthreads();

	if ((threadIdx.x == 0 || idX == 0) && sumVal[idY] > 0)
		atomicAdd(&depthMat[depth + idY * MAX_DEPTH], sumVal[idY]);

	//if (childMat[idX + idY * totalNodes])
	//	atomicAdd(&depthMat[depth + idY * MAX_DEPTH], 1);
	frontMat[idX + idY * totalRef] = childMat[idX + idY * totalRef];
	childMat[idX + idY * totalRef] = false;
	//seqFront[idX + idY * totalNodes] = -1;
}

__global__ void cuda_extract_forest(int* dresults, float* eresults, int* depthMat, Point_Struct* nodeList, int totalNodes)
{
	unsigned depth, ntree;
	depth = threadIdx.x;
	ntree = blockIdx.x;
	__shared__ int dval, total;
	__shared__ float eval;
	if (depth == 0)
	{
		dval = 0;
		eval = 0;
		total = 0;
	}
	__syncthreads();

	if (depthMat[depth + ntree*MAX_DEPTH] > 0)// && nodeList[ntree].isQ)
	{
		atomicAdd(&total, depthMat[depth + ntree*MAX_DEPTH]);
		atomicAdd(&dval, 1);
	}
	__syncthreads();

	if (depthMat[depth + ntree*MAX_DEPTH] > 0)// && nodeList[ntree].isQ)
	{
		float p;
		p = (float)depthMat[depth + ntree*MAX_DEPTH] / (float)total;
		atomicAdd(&eval, -p*log2(p));
	}
	__syncthreads();

	if (depth == 0)
		dresults[ntree] = dval;
	else if (depth == 1)
		eresults[ntree] = eval;

}

__global__ void cuda_calc_degree(int* kresults, char* adjMat, Point_Struct* nodeList, int totalNodes, int totalRef, int totalQuery, int refOnly)
{
	unsigned tid, idX, idY;
	tid = return_thread_index();
	idX = tid % totalRef;
	idY = tid / totalRef;
	extern __shared__ int sumVal1[];

	if (idY >= totalQuery)
		return;
	if (idY == 0)
	{
		kresults[idX] = 0;
	}

	// Init shared memory (only where needed)
	if (sumVal1[idY] != 0)
		sumVal1[idY] = 0;
	__syncthreads();

	if (!adjMat[idX + (idY+refOnly) * totalNodes])// && nodeList[idX].isRef) // 0 is connected
		atomicAdd(&sumVal1[idY], 1);
	__syncthreads();

	if (sumVal1[idY] > 0 && ((threadIdx.x + threadIdx.y) == 0 || idX == 0))
		atomicAdd(&kresults[idY], sumVal1[idY]);
}

__global__ void reduced_add_int(int* g_idata, int* g_odata, int totalNodes)
{
	extern __shared__ int sdata[];

	// Read chuncks into shared memory of each block
	unsigned tid = threadIdx.x + threadIdx.y * blockDim.x;
	unsigned i = return_thread_index();
	if (i < totalNodes)
		sdata[tid] = g_idata[i];
	else
		sdata[tid] = 0;
	__syncthreads();

	// Do reduction
	for (unsigned s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	// Write resutls
	if (tid == 0)
		atomicAdd(g_odata, sdata[0]);
}

__global__ void reduced_add_float(float* g_idata, float* g_odata, int totalNodes)
{
	extern __shared__ float sdata1[];

	// Read chuncks into shared memory of each block
	unsigned tid = threadIdx.x + threadIdx.y * blockDim.x;
	unsigned i = return_thread_index();
	if (i < totalNodes)
		sdata1[tid] = g_idata[i];
	else
		sdata1[tid] = 0;
	__syncthreads();

	// Do reduction
	for (unsigned s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			sdata1[tid] += sdata1[tid + s];
		}
		__syncthreads();
	}

	// Write resutls
	if (tid == 0)
		atomicAdd(g_odata, sdata1[0]);
}

/**************GPU VISIBILITY GRAPH********************/

gpuVG::~gpuVG()
{
	clear();
}

void gpuVG::clear()
{
	//Free memory
	hipFree(_adjMatrix);
	hipFree(_nodeList);
	hipFree(_obsList);
	hipFree(_depthMat);
	hipFree(_parentMat);
	hipFree(_childMat);
	hipFree(_frontMat);
	hipFree(_kresults);
	hipFree(_dresults);
	hipFree(_eresults);
	hipFree(_output);
	_isInit = false;
}

void gpuVG::init(vector<Point_Struct> nodeList, unsigned obsCount, unsigned refCount, unsigned queryCount, unsigned refOnlyCount)
{
	if (_isInit)
		return;

	int totalNodes = nodeList.size();
	_totalNodes = totalNodes;
	_totalQuery = queryCount;
	_totalRef = refCount;
	_refOnly = refOnlyCount;

	// Visibility Grid
	hipMalloc((void**)&_nodeList, totalNodes*sizeof(Point_Struct));
	hipMemcpy(_nodeList, &nodeList[0], totalNodes*sizeof(Point_Struct), hipMemcpyHostToDevice);

	//Initialize adjacency matrix to 0
	hipMalloc((void**)&_adjMatrix, totalNodes*totalNodes*sizeof(char));
	hipMalloc((void**)&_parentMat, _totalQuery*_totalRef*sizeof(bool));
	hipMalloc((void**)&_childMat, _totalQuery*_totalRef*sizeof(bool));
	hipMalloc((void**)&_frontMat, _totalQuery*_totalRef*sizeof(bool));
	hipMalloc((void**)&_seqFront, _totalQuery*_totalRef*sizeof(int));
	hipMemset(_adjMatrix, 0, totalNodes*totalNodes*sizeof(bool));

	// Initialize depth container
	hipMalloc((void**)&_depthMat, _totalQuery*MAX_DEPTH*sizeof(int));

	// Init results
	hipMalloc((void**)&_kresults, _totalQuery*sizeof(int));
	hipMalloc((void**)&_dresults, _totalQuery*sizeof(int));
	hipMalloc((void**)&_eresults, _totalQuery*sizeof(float));
	hipMalloc((void**)&_output, 3 * sizeof(int));

	// MEMORY LEAK DANGER IN CASE OBSTABLE NUMBER IS NOT CONSTANT
	hipMalloc((void**)&_obsList, obsCount*sizeof(Obstacle_Struct));

	_isInit = true;
}

void gpuVG::generate_graph(vector<Obstacle_Struct> obstacles, char obsType)
{
	// Setup block and grid size
	int blockLength = sqrt((double)BLOCK_SIZE);
	int gridLength = ceil(sqrt((double)_totalNodes*_totalNodes / ((double)BLOCK_SIZE)));
	int gridLengthRQ = ceil(sqrt((double)_totalRef*_totalQuery / ((double)BLOCK_SIZE)));
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, gridLength, 1);
	dim3 blocks_rq(gridLengthRQ, gridLengthRQ, 1);

	//Set obstacles to GPU memory
	Obstacle_Struct* obstacle_ptr = &obstacles[0];
	int numObstacles = obstacles.size();
	hipMemcpy(_obsList, obstacle_ptr, obstacles.size()*sizeof(Obstacle_Struct), hipMemcpyHostToDevice);

	// Run kernel
	cuda_generate_graph_mv1 << < blocks, threads >> >(_nodeList, _adjMatrix, _totalNodes, _obsList, obstacles.size(), obsType);

	// Calculate degree
	cuda_calc_degree << <blocks_rq, threads, _totalQuery*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes, _totalRef, _totalQuery, _refOnly);

	// *****************SYNC
	hipDeviceSynchronize();
}

char* gpuVG::get_adjMatrix()
{
	//Copy memory back to host
	char* adjMatrix_ptr = (char*)malloc(_totalNodes*_totalNodes*sizeof(char));
	hipMemcpy(adjMatrix_ptr, _adjMatrix, _totalNodes*_totalNodes*sizeof(char), hipMemcpyDeviceToHost);
	return adjMatrix_ptr;
}

int* gpuVG::get_depthMatrix()
{
	//Copy memory back to host
	int* depthMatrix_ptr = (int*)malloc(_totalQuery*MAX_DEPTH*sizeof(int));
	hipMemcpy(depthMatrix_ptr, _depthMat, _totalQuery*MAX_DEPTH*sizeof(int), hipMemcpyDeviceToHost);
	return depthMatrix_ptr;
}

int* gpuVG::get_degree()
{
	int blockLength = ceil(sqrt((double)BLOCK_SIZE));
	int gridLength = ceil((double)_totalQuery / BLOCK_SIZE);
	int gridLengthX = ceil((double)_totalNodes*_totalNodes / BLOCK_SIZE);
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);

	// Initialize output
	hipMemset(_output, 0, sizeof(int));

	// Calculate degree
	//cuda_calc_degree << <blocks_2, threads, _totalNodes*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes);

	// Take sum over nodes
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_kresults, &_output[0], _totalNodes);
	int* output = (int*)malloc(sizeof(int));
	hipMemcpy(output, _output, sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

void gpuVG::generate_forest()
{
	// Setup block and grid size
	int blockLength = ceil(sqrt((double)BLOCK_SIZE));
	int gridLengthX = ceil((double)_totalNodes*_totalNodes / BLOCK_SIZE);
	int gridLengthY = ceil((double)_totalNodes / BLOCK_SIZE);
	int gridLengthQ = ceil((double)_totalQuery / BLOCK_SIZE);
	int gridLengthR = ceil((double)_totalRef / BLOCK_SIZE);
	int gridLengthRQ = ceil((double)_totalRef*_totalQuery / BLOCK_SIZE);
	dim3 threads(BLOCK_SIZE, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);
	dim3 blocks_1(gridLengthY, 1, 1);
	dim3 blocks_q(gridLengthQ, 1, 1);
	dim3 blocks_rq(gridLengthRQ, 1, 1);
	dim3 blocks_3q(gridLengthR, _totalQuery, 1);
	dim3 blocks_3(gridLengthY, _totalNodes, 1);
	hipError_t err;

	// Init tree
	hipMemset(_childMat, false, _totalQuery*_totalRef*sizeof(bool));
	hipMemset(_parentMat, false, _totalQuery*_totalRef*sizeof(bool));
	hipMemset(_depthMat, 0, _totalQuery*MAX_DEPTH*sizeof(int));
	//cuda_init_forest << < blocks_2, threads >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, _totalNodes);
	cuda_init_forest << < blocks_q, threads >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, _totalRef, _totalQuery, _refOnly);

	//int* res = (int*)malloc(_totalNodes*_totalNodes*sizeof(int));
	for (int i = 0; i < MAX_DEPTH; i++)
	{
		// Run kernel
		//cuda_expand_forest << < blocks_3, threads, _totalNodes*sizeof(int) >> >(_parentMat, _childMat, _seqFront, _depthMat, _nodeList, _adjMatrix, i, _totalNodes);
		cuda_expand_forest << < blocks_3q, threads >> >(_parentMat, _childMat, _seqFront, _depthMat, _nodeList, _adjMatrix, i, _totalNodes, _totalRef, _totalQuery);

		// Update tree
		//cuda_update_forest << < blocks_2, threads, _totalNodes*sizeof(int) >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, i, _totalRef, _totalQuery);
		cuda_update_forest << < blocks_rq, threads, _totalQuery*sizeof(int) >> >(_parentMat, _childMat, _frontMat, _seqFront, _depthMat, i, _totalRef, _totalQuery);

		// Make front list
		if (i < MAX_DEPTH - 1)
			cuda_sequence_list << < blocks_q, threads >> >(_frontMat, _seqFront, _totalRef, _totalQuery);
	}
	dim3 threadsD(MAX_DEPTH, 1, 1);
	dim3 blocksD(_totalQuery, 1, 1);

	// Calculate depth and entropy
	cuda_extract_forest << <blocksD, threadsD >> >(_dresults, _eresults, _depthMat, _nodeList, _totalNodes);
}

int* gpuVG::get_output()
{
	int blockLength = ceil(sqrt((double)BLOCK_SIZE));
	int gridLength = ceil((double)_totalQuery / BLOCK_SIZE);
	int gridLengthX = ceil((double)_totalNodes*_totalNodes / BLOCK_SIZE);
	dim3 threads(blockLength, blockLength, 1);
	dim3 blocks(gridLength, 1, 1);
	dim3 blocks_2(gridLengthX, 1, 1);

	// Initialize output
	hipMemset(_output, 0, 3 * sizeof(int));

	// Calculate degree
	//cuda_calc_degree << <blocks_2, threads, _totalNodes*sizeof(int) >> >(_kresults, _adjMatrix, _nodeList, _totalNodes);

	// Take sum over nodes
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_kresults, &_output[0], _totalQuery);
	reduced_add_int << <blocks, threads, BLOCK_SIZE*sizeof(int) >> >(_dresults, &_output[1], _totalQuery);
	reduced_add_float << <blocks, threads, BLOCK_SIZE*sizeof(float) >> >(_eresults, (float*)&_output[2], _totalQuery);
	int* output = (int*)malloc(3 * sizeof(int));
	hipMemcpy(output, _output, 3 * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

int* gpuVG::get_nodesDegree()
{
	int* output = (int*)malloc(_totalQuery * sizeof(int));
	hipMemcpy(output, _kresults, _totalQuery * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

int* gpuVG::get_nodesDepth()
{
	int* output = (int*)malloc(_totalQuery * sizeof(int));
	hipMemcpy(output, _dresults, _totalQuery * sizeof(int), hipMemcpyDeviceToHost);
	return output;
}

float* gpuVG::get_nodesEntropy()
{
	float* output = (float*)malloc(_totalQuery * sizeof(float));
	hipMemcpy(output, _eresults, _totalQuery * sizeof(float), hipMemcpyDeviceToHost);
	return output;
}
